#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <cmath>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include "mpi.h"

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

#define _i(i, j, k) ((k + 1) * (n_x + 2) * (n_y + 2) + (j + 1) * (n_x + 2) + i + 1)
#define _ib(i, j, k) (k * nb_x * nb_y + j * nb_x + i)


__global__ void kernalCompute(double *data, double *next, int n_x, int n_y, int n_z, double hx, double hy, double hz) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_x * n_y * n_z;

	int i, j, k;

	for(int l = idx; l < size; l += offset)	{
		i = l % n_x;
		j = (l / n_x ) % n_y;
		k = l / (n_x * n_y);
		next[_i(i, j, k)] = 0.5 * ((data[_i(i + 1, j, k)] + data[_i(i - 1, j, k)]) / (hx * hx) +
											(data[_i(i, j + 1, k)] + data[_i(i, j - 1, k)]) / (hy * hy) +
											(data[_i(i, j, k + 1)] + data[_i(i, j, k - 1)]) / (hz * hz)) /
												(1.0 / (hx * hx) + 1.0 / (hy * hy) + 1.0 / (hz * hz));
	}
}



__global__ void kernalComputeDiff(double *data, double *next, int n_x, int n_y, int n_z) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = (n_x + 2) * (n_y + 2) * (n_z + 2);
	int i, j, k;

	for (int l = idx; l < size; l += offset)	{
		i = l % (n_x + 2);
		j = (l / (n_x + 2)) % (n_y + 2);
		k = l / ((n_x + 2) * (n_y + 2));
		next[l] = fabs(next[l] - data[l]);
		if (i == 0 || j == 0 || k == 0 || i == (n_x + 1) || j == (n_y + 1) || k == (n_z + 1)) {
			next[l] = 0;
		}
	}
}


__global__ void kernalIJNt_to_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_x * n_y;

	int i, j;

	for(int l = idx; l < size; l += offset)	{
		i = l % n_x;
		j = l / n_x;
		buff[i * n_y + j] = data[_i(i, j, layer)];
	}
}

__global__ void kernalIJNt_from_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_x * n_y;

	int i, j;

	for(int l = idx; l < size; l += offset)	{
		i = l % n_x;
		j = l / n_x;
		data[_i(i, j, layer)] = buff[i * n_y + j];
	}
}

__global__ void kernalINK_to_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_x * n_z;

	int i, k;

	for(int l = idx; l < size; l += offset)	{
		i = l % n_x;
		k = l / n_x;
		buff[i * n_z + k] = data[_i(i, layer, k)];
	}
}

__global__ void kernalINK_from_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_x * n_z;

	int i, k;

	for(int l = idx; l < size; l += offset)	{
		i = l % n_x;
		k = l / n_x;
		data[_i(i, layer, k)] = buff[i * n_z + k];
	}
}

__global__ void kernalNJK_to_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_y * n_z;

	int j, k;

	for(int l = idx; l < size; l += offset)	{
		j = l % n_y;
		k = l / n_y;
		buff[j * n_z + k] = data[_i(layer, j, k)];
	}
}

__global__ void kernalNJK_from_gbuff(double *data, double *buff, int n_x, int n_y, int n_z, int layer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	int size = n_y * n_z;

	int j, k;

	for(int l = idx; l < size; l += offset)	{
		j = l % n_y;
		k = l / n_y;
		data[_i(layer, j, k)] = buff[j * n_z + k];
	}
}


int main(int argc, char *argv[]) {
	int ib, jb, kb, nb_x, nb_y, nb_z, n_x, n_y, n_z;
	int i, j, k;
	int id, numproc, proc_name_len, gpu_count;
	char proc_name[MPI_MAX_PROCESSOR_NAME];
	char file_path[256] = "mpi.out";

	double lx, ly, lz, hx, hy, hz, bc_down, bc_up, bc_left, bc_right, bc_back, bc_front;
	double *data, *temp, *buffs, *buffr;
	double u_0, max_dif_l, max_dif, eps;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);
	MPI_Get_processor_name(proc_name, &proc_name_len);

	CSC(hipGetDeviceCount(&gpu_count));
	CSC(hipSetDevice(id % gpu_count));

	MPI_Barrier(MPI_COMM_WORLD);

	if (id == 0) {
		scanf("%d%d%d",&nb_x,&nb_y,&nb_z );
		scanf("%d%d%d",&n_x,&n_y,&n_z );
		scanf("%s", file_path);
		scanf("%lf",&eps);
		scanf("%lf%lf%lf",&lx,&ly,&lz);
		scanf("%lf%lf%lf%lf%lf%lf",&bc_front,&bc_back, &bc_left, &bc_right, &bc_down, &bc_up);
		scanf("%lf",&u_0);
	}


	MPI_Bcast(&n_x, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&n_y, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&n_z, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&nb_x, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&nb_y, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&nb_z, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lz, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_down, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_up, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_left, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_right, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_back, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&bc_front, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&u_0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);



	ib = id % nb_x;
	jb = (id / nb_x) % nb_y;
	kb = id / (nb_x * nb_y);

	hx = lx / (double)(n_x * nb_x);
	hy = ly / (double)(n_y * nb_y);
	hz = lz / (double)(n_z * nb_z);

	data = (double *)malloc(sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2));

	double *dev_data, *dev_next, *dev_buff;
	CSC(hipMalloc(&dev_data, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2)));
	CSC(hipMalloc(&dev_next, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2)));

	thrust::device_ptr<double> data_tr;

	int n_max1 = std::max(n_x, n_y);
	int n_max2 = std::max(n_z, n_x + n_y - n_max1);

	buffs = (double *)malloc(sizeof(double) * (n_max1 + 2) * (n_max2 + 2));
	buffr = (double *)malloc(sizeof(double) * (n_max1 + 2) * (n_max2 + 2));

	CSC(hipMalloc(&dev_buff, sizeof(double) * (n_max1 + 2) * (n_max2 + 2)));


	if (id == 0) {
		fprintf(stderr, "%f %f %f %f %f %f %f %f %f %f %f %f %f\n", lx, ly, lz, hx, hy, hz, bc_front, bc_back, bc_left, bc_right, bc_down, bc_up, u_0);
		fprintf(stderr, "%f %d %d %d %d %d %d\n", eps, nb_x, nb_y, nb_z, n_x, n_y, n_z);
		fflush(stderr);
	}

	for(i = 0; i < n_x; i++)					// Инициализация блока
		for(j = 0; j < n_y; j++)
			for(k = 0; k < n_z; k++)
				data[_i(i, j, k)] = u_0;

	for(i = -1; i <= n_x; i++) {
		for(j = -1; j <= n_y; j++) {
			data[_i(i, j, -1)] = bc_front;
			data[_i(i, j, n_z)] = bc_back;
		}
	}
	for(i = -1; i <= n_x; i++) {
		for(k = -1; k <= n_z; k++) {
			data[_i(i, -1, k)] = bc_down;
			data[_i(i, n_y, k)] = bc_up;
		}
	}
	for(j = -1; j <= n_y; j++) {
		for(k = -1; k <= n_z; k++) {
			data[_i(-1, j, k)] = bc_left;
			data[_i(n_x, j, k)] = bc_right;
		}
	}

	CSC(hipMemcpy(dev_data, data, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2), hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_next, data, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2), hipMemcpyHostToDevice));


	int it = 0;
	while(1) {
		MPI_Barrier(MPI_COMM_WORLD);
		it++;
		for(int dir = 0; dir < 2; dir++) {
			if ((ib + dir) & 1) {
				if (ib > 0) {

					kernalNJK_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, 0);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_y * n_z, MPI_DOUBLE, _ib(ib - 1, jb, kb), id,
								buffr, n_y * n_z, MPI_DOUBLE, _ib(ib - 1, jb, kb), _ib(ib - 1, jb, kb), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalNJK_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, -1);

				}
			} else {
				if (ib + 1 < nb_x) {

					kernalNJK_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_x - 1);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_y * n_z, MPI_DOUBLE, _ib(ib + 1, jb, kb), id,
								buffr, n_y * n_z, MPI_DOUBLE, _ib(ib + 1, jb, kb), _ib(ib + 1, jb, kb), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalNJK_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_x);

				}
			}
		}
		// Y-axies (j)
		for(int dir = 0; dir < 2; dir++) {
			if ((jb + dir)& 1) {
				if (jb > 0) {
					kernalINK_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, 0);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_x * n_z, MPI_DOUBLE, _ib(ib, jb - 1, kb), id,
								buffr, n_x * n_z, MPI_DOUBLE, _ib(ib, jb - 1, kb), _ib(ib, jb - 1, kb), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalINK_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, -1);
				}
			} else {
				if (jb + 1 < nb_y) {
					kernalINK_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_y - 1);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_x * n_z, MPI_DOUBLE, _ib(ib, jb + 1, kb), id,
								buffr, n_x * n_z, MPI_DOUBLE, _ib(ib, jb + 1, kb), _ib(ib, jb + 1, kb), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalINK_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_y);
				}
			}
		}

		// Z-axies (k)
		for(int dir = 0; dir < 2; dir++) {
			if ((kb + dir)& 1) {
				if (kb > 0) {
					kernalIJNt_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, 0);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_x * n_y, MPI_DOUBLE, _ib(ib, jb, kb - 1), id,
								buffr, n_x * n_y, MPI_DOUBLE, _ib(ib, jb, kb - 1), _ib(ib, jb, kb - 1), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalIJNt_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, -1);
				}
			} else {
				if (kb + 1 < nb_z) {
					kernalIJNt_to_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_z - 1);
					CSC(hipMemcpy(buffs, dev_buff, sizeof(double) * n_x * n_y, hipMemcpyDeviceToHost));

					MPI_Sendrecv(buffs, n_x * n_y, MPI_DOUBLE, _ib(ib, jb, kb + 1), id,
								buffr, n_x * n_y, MPI_DOUBLE, _ib(ib, jb, kb + 1), _ib(ib, jb, kb + 1), MPI_COMM_WORLD, &status);

					CSC(hipMemcpy(dev_buff, buffr, sizeof(double) * n_x * n_y, hipMemcpyHostToDevice));
					kernalIJNt_from_gbuff<<<64, 64>>>(dev_data, dev_buff, n_x, n_y, n_z, n_z);
				}
			}
		}


		MPI_Barrier(MPI_COMM_WORLD);

		kernalCompute<<<64, 64>>>(dev_data, dev_next, n_x, n_y, n_z, hx, hy, hz);
		CSC(hipGetLastError());

		temp = dev_next;
		dev_next = dev_data;
		dev_data = temp;

		max_dif_l = 0;

		kernalComputeDiff<<<64, 64>>>(dev_data, dev_next, n_x, n_y, n_z);
		CSC(hipGetLastError());

		data_tr = thrust::device_pointer_cast(dev_next);

		max_dif_l = *(thrust::max_element(data_tr, data_tr + (n_x + 2) * (n_y + 2) * (n_z + 2)));

		CSC(hipMemcpy(dev_next, dev_data, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2), hipMemcpyDeviceToDevice));

		MPI_Allreduce(&max_dif_l, &max_dif, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

		if (max_dif < eps) {
			fprintf(stderr, "e %f, max_dif_l %f, max_dif%f, it%d\n", eps, max_dif_l, max_dif, it);
			fflush(stderr);
			break;
		}
	}

	CSC(hipMemcpy(data, dev_data, sizeof(double) * (n_x + 2) * (n_y + 2) * (n_z + 2), hipMemcpyDeviceToHost));

	CSC(hipFree(dev_data));
	CSC(hipFree(dev_next));
	CSC(hipFree(dev_buff));

	int print_size = 15;
	char* str = (char *)malloc(sizeof(char) * (n_x * n_y * n_z * print_size + 1));
	memset(str, ' ', sizeof(char) * (n_x * n_y * n_z * print_size));

	for(k = 0; k < n_z; k++) {
		for(j = 0; j < n_y; j++) {
			for(i = 0; i < n_x; i++)
				sprintf(str + (i + j * n_x + k * n_x * n_y) * print_size, "%.7e ", data[_i(i, j, k)]);
			if (ib + 1 == nb_x)
				str[((j + 1) * n_x + k * n_x * n_y) * print_size - 1] = '\n';
		}
	}

	for (i = 0; i < n_x * n_y * n_z * print_size; i++)
		if (str[i] == '\0')
			str[i] = ' ';
	str[n_x * n_y * n_z * print_size] = '\0';

	MPI_Datatype gridT, blockT;
	MPI_Type_vector(n_y, n_x * print_size, n_x * print_size * nb_x, MPI_CHAR, &blockT);
	MPI_Type_commit(&blockT);

	MPI_Type_create_hvector(n_z, 1, n_x * nb_x * n_y * nb_y * print_size, blockT, &gridT);
	MPI_Type_commit(&gridT);

	MPI_File fp;
	MPI_File_delete(file_path, MPI_INFO_NULL);
	MPI_File_open(MPI_COMM_WORLD, file_path, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);

	MPI_File_set_view(fp, sizeof(char) * (n_x * ib + n_x * nb_x * n_y * jb + n_x * nb_x * n_y * nb_y * n_z * kb) * print_size, MPI_CHAR, gridT, "native", MPI_INFO_NULL);
	MPI_File_write_all(fp, str, n_x * n_y * n_z * print_size, MPI_CHAR, &status);

	MPI_File_close(&fp);

	free(data);
	free(str);
	free(buffr);
	free(buffs);

	MPI_Finalize();
	return 0;
}
