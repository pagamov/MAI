#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/count.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

struct comparator {												
	__host__ __device__ bool operator()(double a, double b) {
		return abs(a) < abs(b);
	}
};

__global__ void LUP_swap(double * A, int i, int n, int newidx) {
	double piv;
	for (int var = blockDim.x * blockIdx.x + threadIdx.x; var < n; var += blockDim.x * gridDim.x) {
		piv = A[newidx + n * var];
        A[newidx + n * var] = A[i + n * var];
        A[i + n * var] = piv;
	}
}

__global__ void LUP_N(double * A, int i, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = blockDim.x * gridDim.x;
	for (int var = idx + i + 1; var < n; var += shift)
		A[var + n * i] /= A[i + n * i];
}

__global__ void LUP(double * A, int i, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int shiftx = blockDim.x * gridDim.x;
	int shifty = blockDim.y * gridDim.y;
	for (int var = idx + i + 1; var < n; var += shiftx)
		for (int k = idy + i + 1; k < n; k += shifty)
			A[var + n * k] -= A[var + n * i] * A[i + n * k];
}

int main() {
    int n; comparator comp;
    cin >> n;
    double * A = (double *)malloc(sizeof(double) * n * n);
	int * newidxarr = (int *)malloc(sizeof(int) * n);
    
    for (int y = 0; y < n; y++)
        for (int x = 0; x < n; x++)
            cin >> A[x * n + y];
            
    double * A_DEV;
    CSC(hipMalloc(&A_DEV, sizeof(double) * n * n));
    CSC(hipMemcpy(A_DEV, A, sizeof(double) * n * n, hipMemcpyHostToDevice));
	
    for(int i = 0; i < n; i++) {
		thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(A_DEV) + (i * n + i);
		thrust::device_ptr<double> max = thrust::max_element(d_ptr, d_ptr + (n - i), comp);
		newidxarr[i] = max - d_ptr + i;
		LUP_swap <<<32,32>>> (A_DEV, i, n, newidxarr[i]);
		LUP_N <<<32,32>>> (A_DEV, i, n);
		LUP <<<dim3(32,32),dim3(32,32)>>> (A_DEV, i, n);
    }
	                      
    CSC(hipMemcpy(A, A_DEV, sizeof(double) * n * n, hipMemcpyDeviceToHost));
    
    for (int y = 0; y < n; y++) {
        for (int x = 0; x < n; x++)
            printf("%.10lf ", A[x * n + y]);
        printf("\n");
    }
    for (int i = 0; i < n; i++)
		printf("%d ", newidxarr[i]);
    printf("\n");
	
	free(newidxarr);
    CSC(hipFree(A_DEV));
    free(A);
    return 0;
}