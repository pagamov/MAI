
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>

using namespace std;

#define SR_S 512
#define ull unsigned long long

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

int pow(int n, int p) {
    int res = 1;
    for (int i = 0; i < p; i++)
        res *= n;
    return res;
}

__global__ void B_shared(int * arr, int size_p, int upd_n, int sign_p) {
	int tmp;
	__shared__ int buf[SR_S];
	for (ull i = (ull)blockIdx.x * SR_S; i < upd_n; i += (ull)gridDim.x * SR_S) {
		for (int k = threadIdx.x; k < SR_S; k += blockDim.x)
			buf[k] = arr[i + k];
		__syncthreads();

		for (ull size_k_p = size_p; size_k_p >= 1; size_k_p--) {
			ull size_k = 1 << size_k_p;
			for (ull j = threadIdx.x; j < SR_S / 2; j += blockDim.x) {
				ull b = (ull)(j >> (size_k_p - 1)) * size_k + (j & ((1 << (size_k_p - 1)) - 1));
				if ((buf[b] > buf[b + (size_k >> 1)]) != (((i + b) / (1 << sign_p)) & 1)) {
					tmp = buf[b];
					buf[b] = buf[b + (size_k >> 1)];
					buf[b + (size_k >> 1)] = tmp;
				}
			}
			__syncthreads();
		}
		for (int k = threadIdx.x; k < SR_S; k += blockDim.x)
			arr[i + k] = buf[k];
		__syncthreads();
	}
}

__global__ void B_global(int * arr, int size_p, int upd_n, int sign_p) {
	int tmp;
	ull size = 1 << size_p;
	for (ull i = (ull)blockIdx.x * size; i < upd_n; i += (ull)gridDim.x * size) {
		for (ull j = threadIdx.x; j < size / 2; j += blockDim.x) {
			if ((arr[i + j] > arr[i + j + size / 2]) == (((i + j) / (1 << sign_p)) % 2 == 0)) {
				tmp = arr[i+j];
				arr[i + j] = arr[i + j + size / 2];
				arr[i + j + size / 2] = tmp;
			}
		}
	}
}

int main() {
    bool verbose = false; // 0 for binary, 1 for normal
    int n, upd_n;

    if (verbose)
        cin >> n;
    else
        fread(&n, 4, 1, stdin);

    int p = 0;
    while (pow(2, p) < n)
        p++;
    upd_n = pow(2, p);

	int * arr = (int *)malloc(4 * upd_n);

    if (verbose)
        for (int i = 0; i < n; i++)
            cin >> arr[i];
    else
        fread(arr, 4, n, stdin);

	for (int i = n; i < upd_n; i++)
		arr[i] = INT_MAX;

	int * dev_arr;
	CSC(hipMalloc(&dev_arr, 4 * upd_n));
	CSC(hipMemcpy(dev_arr, arr, 4 * upd_n, hipMemcpyHostToDevice));

	for (int i = 1; pow(2, i) <= upd_n; i++) {
		for (int j = i; j >= 1; j--) {
			if (j <= 9) {
				B_shared <<<64, 64>>> (dev_arr, j, upd_n, i);
				break;
			} else
				B_global <<<64, 128>>> (dev_arr, j, upd_n, i);
			CSC(hipDeviceSynchronize());
			CSC(hipGetLastError());
		}
	}
	CSC(hipMemcpy(arr, dev_arr, 4 * upd_n, hipMemcpyDeviceToHost));

    if (verbose) {
        for (int i = 0; i < n; i++)
            cout << arr[i] << " ";
        cout << endl;
    } else {
        fwrite(arr, 4, n, stdout);
    }

    CSC(hipFree(dev_arr));
	free(arr);
	return 0;
}
