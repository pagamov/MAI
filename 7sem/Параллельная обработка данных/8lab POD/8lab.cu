#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <math.h>
#include <mpi.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

using namespace std;

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)

#define _i(i, j, k) ((k + 1) * (nx + 2) * (ny + 2) + (j + 1) * (nx + 2) + i + 1)
#define _iD(i, j, k) ((k + 1) * (dim[0] + 2) * (dim[1] + 2) + (j + 1) * (dim[0] + 2) + i + 1)
#define _ib(i, j, k) (k * box[0] * box[1] + j * box[0] + i)

__global__ void kernel_setup(double * arr, int nx, int ny, int nz, double u_0) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < nz; k += blockDim.z * gridDim.z)
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j += blockDim.x * gridDim.x)
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    			arr[_i(i, j, k)] = u_0;
}

__global__ void kernel_LR_setup(double * arr, int nx, int ny, int nz, double bc, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j += blockDim.x * gridDim.x)
    		arr[_i(x_ind, j, k)] = bc;
}

__global__ void kernel_FB_setup(double * arr, int nx, int ny, int nz, double bc, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		arr[_i(i, y_ind, k)] = bc;
}

__global__ void kernel_DU_setup(double * arr, int nx, int ny, double bc, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		arr[_i(i, j, z_ind)] = bc;
}

__global__ void kernel_send_LR(double * buf, double* arr, int nx, int ny, int nz, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j += blockDim.x * gridDim.x)
    		buf[k * ny + j] = arr[_i(x_ind, j, k)];
}

__global__ void kernel_recive_LR(double * buf, double * arr, int nx, int ny, int nz, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j += blockDim.x * gridDim.x)
    		arr[_i(x_ind, j, k)] = buf[k * ny + j];
}

__global__ void kernel_send_FB(double * buf, double * arr, int nx, int ny, int nz, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		buf[k * nx + i] = arr[_i(i, y_ind, k)];
}

__global__ void kernel_recive_FB(double * buf, double * arr, int nx, int ny, int nz, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		arr[_i(i, y_ind, k)] = buf[k * nx + i];
}

__global__ void kernel_send_DU(double * buf, double * arr, int nx, int ny, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		buf[j * nx + i] = arr[_i(i, j, z_ind)];
}

__global__ void kernel_recive_DU(double * buf, double * arr, int nx, int ny, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j += blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x)
    		arr[_i(i, j, z_ind)] = buf[j * nx + i];
}

__global__ void kernel_compute(double * arr, double * next, int nx, int ny, int nz, double hx, double hy, double hz) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < nz; k += blockDim.z * gridDim.z) {
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j += blockDim.y * gridDim.y) {
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i += blockDim.x * gridDim.x) {
    			next[_i(i, j, k)] = 0.5 * ((arr[_i(i + 1, j, k)] + arr[_i(i - 1, j, k)]) / (hx * hx) +
						(arr[_i(i, j + 1, k)] + arr[_i(i, j - 1, k)]) / (hy * hy) +
						(arr[_i(i, j, k + 1)] + arr[_i(i, j, k - 1)]) / (hz * hz)) /
						(1.0 / (hx * hx) + 1.0 / (hy * hy) + 1.0 / (hz * hz));
    		}
    	}
    }
}

__global__ void kernel_diff(double * arr, double * next, int nx, int ny, int nz) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z - 1; k <= nz; k += blockDim.z * gridDim.z)
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y - 1; j <= ny; j += blockDim.y * gridDim.y)
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x - 1; i <= nx; i += blockDim.x * gridDim.x)
    			arr[_i(i, j, k)] = ((i != -1) && (j != -1) && (k != -1) && (i != nx) && (j != ny) && (k != nz)) * fabs(next[_i(i, j, k)] - arr[_i(i, j, k)]);
}


int main(int argc, char* argv[]) {
    int id, numproc;
	int ib, jb, kb;
    int box[3];
    int dim[3];
    char filename[100];
    double l[3];
	double hx, hy, hz;
    double u[6], u_0;
	double eps, diff;
	double *temp, *buff;
    bool f = true;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	int deviceCount;
	CSC(hipGetDeviceCount(&deviceCount));
	CSC(hipSetDevice(id % deviceCount));

	if (id == 0) {
        cin >> box[0] >> box[1] >> box[2];
        cin >> dim[0] >> dim[1] >> dim[2];
		scanf("%s", filename);
		cin >> eps;
        cin >> l[0] >> l[1] >> l[2];
        cin >> u[4] >> u[5] >> u[0] >> u[1] >> u[2] >> u[3] >> u_0;
	}

    MPI_Bcast(dim, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(box, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(l, 3, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(u, 6, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(&u_0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(filename, 100, MPI_CHAR, 0, MPI_COMM_WORLD);

    ib = id % (box[0] * box[1]) % box[0];
    jb = id % (box[0] * box[1]) / box[0];
	kb = id / (box[0] * box[1]);

	hx = l[0] / (dim[0] * box[0]);
	hy = l[1] / (dim[1] * box[1]);
	hz = l[2] / (dim[2] * box[2]);

	double * dev_data;
	double * dev_next;
	CSC(hipMalloc(&dev_data, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2)));
	CSC(hipMalloc(&dev_next, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2)));

	buff = (double *)malloc(sizeof(double) * max(dim[0], dim[1]) * max(dim[1], dim[2]));
	int buffer_size;
	MPI_Pack_size(max(dim[0], dim[1]) * max(dim[1], dim[2]), MPI_DOUBLE, MPI_COMM_WORLD, &buffer_size);
	buffer_size = 6 * (buffer_size + MPI_BSEND_OVERHEAD);
	double * buffer = (double*)malloc(buffer_size);
	MPI_Buffer_attach(buffer, buffer_size);
    double * allgbuff = (double *)malloc(sizeof(double) * box[0] * box[1] * box[2]);
	double * dev_buff;
	CSC(hipMalloc(&dev_buff, sizeof(double) * max(dim[0], dim[1]) * max(dim[1], dim[2])));

	kernel_setup<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dim[0], dim[1], dim[2], u_0);
	CSC(hipGetLastError());

	while (f) {
		MPI_Barrier(MPI_COMM_WORLD);

		if (ib + 1 < box[0]) {
			kernel_send_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[0]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[1] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib + 1, jb, kb), id, MPI_COMM_WORLD);
		}

		if (jb + 1 < box[1]) {
			kernel_send_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[1]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb + 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb + 1 < box[2]) {
			kernel_send_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[1], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb + 1), id, MPI_COMM_WORLD);
		}

		if (ib > 0) {
			kernel_send_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[1] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib - 1, jb, kb), id, MPI_COMM_WORLD);
		}

		if (jb > 0) {
			kernel_send_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb - 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb > 0) {
			kernel_send_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[1], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb - 1), id, MPI_COMM_WORLD);
		}
        
        MPI_Barrier(MPI_COMM_WORLD);

		if (ib > 0) {
			MPI_Recv(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib - 1, jb, kb), _ib(ib - 1, jb, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[1] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], -1);
			CSC(hipGetLastError());
		} else {
			kernel_LR_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], u[0], -1);
			CSC(hipGetLastError());
		}

		if (jb > 0) {
			MPI_Recv(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb - 1, kb), _ib(ib, jb - 1, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], -1);
			CSC(hipGetLastError());
		} else {
			kernel_FB_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], u[2], -1);
			CSC(hipGetLastError());
		}

		if (kb > 0) {
			MPI_Recv(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb - 1), _ib(ib, jb, kb - 1), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[1], hipMemcpyHostToDevice));
			kernel_recive_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], -1);
			CSC(hipGetLastError());
		} else {
			kernel_DU_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], u[4], -1);
			CSC(hipGetLastError());
		}

		if (ib + 1 < box[0]) {
			MPI_Recv(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib + 1, jb, kb), _ib(ib + 1, jb, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[1] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[0]);
			CSC(hipGetLastError());
		} else {
			kernel_LR_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], u[1], dim[0]);
			CSC(hipGetLastError());
		}

		if (jb + 1 < box[1]) {
			MPI_Recv(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb + 1, kb), _ib(ib, jb + 1, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[1]);
			CSC(hipGetLastError());
		} else {
			kernel_FB_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], u[3], dim[1]);
			CSC(hipGetLastError());
		}

		if (kb + 1 < box[2]) {
			MPI_Recv(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb + 1), _ib(ib, jb, kb + 1), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[1], hipMemcpyHostToDevice));
			kernel_recive_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2]);
			CSC(hipGetLastError());
		} else {
			kernel_DU_setup<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], u[5], dim[2]);
			CSC(hipGetLastError());
		}

		MPI_Barrier(MPI_COMM_WORLD);
		diff = 0.0;

		kernel_compute<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dev_next, dim[0], dim[1], dim[2], hx, hy, hz);
		CSC(hipGetLastError());

		kernel_diff<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dev_next, dim[0], dim[1], dim[2]);
		CSC(hipGetLastError());

		thrust::device_ptr<double> diffs = thrust::device_pointer_cast(dev_data);
        thrust::device_ptr<double> max_eps = thrust::max_element(diffs, diffs + (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2));
        diff = *max_eps;

        MPI_Allgather(&diff, 1, MPI_DOUBLE, allgbuff, 1, MPI_DOUBLE, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
        f = false;
        for (int i = 0; i < box[0] * box[1] * box[2]; i++)
            if (allgbuff[i] > eps)
                f = true;

		temp = dev_next;
		dev_next = dev_data;
		dev_data = temp;
	}

	double * data = (double *)malloc(sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2));
	hipMemcpy(data, dev_data, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2), hipMemcpyDeviceToHost);

	char * bf = (char *)malloc(sizeof(char) * dim[0] * dim[1] * dim[2] * 14);
	memset(bf, ' ', sizeof(char) * dim[0] * dim[1] * dim[2] * 14);

	for (int k = 0; k < dim[2]; k++)
		for (int j = 0; j < dim[1]; j++)
			for (int i = 0; i < dim[0]; i++)
				sprintf(bf + (k * dim[0] * dim[1] + j * dim[0] + i) * 14, "%.6e", data[_iD(i, j, k)]);

	for (int i = 0; i < dim[0] * dim[1] * dim[2] * 14; i++)
		if (bf[i] == '\0')
			bf[i] = ' ';

	MPI_File fp;
	MPI_Datatype filetype;
	int sizes[] = {dim[2] * box[2], dim[1] * box[1], dim[0] * box[0] * 14};
	int subsizes[] = {dim[2], dim[1], dim[0] * 14};
	int starts[] = {id / (box[0] * box[1]) * dim[2], id % (box[0] * box[1]) / box[0] * dim[1], id % (box[0] * box[1]) % box[0] * dim[0] * 14};
	MPI_Type_create_subarray(3, sizes, subsizes, starts, MPI_ORDER_C, MPI_CHAR, &filetype);
	MPI_Type_commit(&filetype);

	MPI_File_delete(filename, MPI_INFO_NULL);
	MPI_File_open(MPI_COMM_WORLD, filename, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);
	MPI_File_set_view(fp, 0, MPI_CHAR, filetype, "native", MPI_INFO_NULL);
	MPI_File_write_all(fp, bf, dim[0] * dim[1] * dim[2] * 14, MPI_CHAR, MPI_STATUS_IGNORE);
	MPI_File_close(&fp);

	MPI_Type_free(&filetype);
	MPI_Finalize();
    CSC(hipFree(dev_data));
	CSC(hipFree(dev_next));
	CSC(hipFree(dev_buff));
    // free(allgbuff);
	free(data);
	free(buff);
	free(bf);
	return 0;
}
