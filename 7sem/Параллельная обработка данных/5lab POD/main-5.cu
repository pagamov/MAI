#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <limits>


#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)


__device__ void swap_GPU(int& a, int& b) {
	int tmp = a;
	a = b;
	b = tmp;
}

__device__ void print_data(int* data, int N) {
	for(int i = 0; i < N; i++) {
		printf("%d ", data[i]);
	}
	printf("\n");
}

void swap(int& a, int& b) {
	int tmp = a;
	a = b;
	b = tmp;
}

void B(int* data, int size, int N, int sign_shift) {	
	for(int i = 0; i < N; i+=size) {
		for(int j = 0; j < size / 2; j++) {
			if ((data[i+j] > data[i+j + size / 2]) == (((i+j) / sign_shift) % 2 == 0)) {
				swap(data[i+j], data[i+j + size / 2]);
			}
		}
	}
}

// __global__ void  B_shared_kernal(int* data, int size_p, int N, int sign_shift_p) {
// 	int sign_shift = 1<< sign_shift_p;
// 	int size = 1<<size_p;
// 	__shared__ int buff[512];


// 	for(unsigned long long i = blockIdx.x * size; i < N; i += gridDim.x * size) {
// 		// memcpy(&data[i], buff, sizeof(int) * size);

// 		for(int k = threadIdx.x; k < size; k += blockDim.x)
// 			buff[k] = data[i + k];
// 		__syncthreads();
// 		// printf("i = %d\n", i);
// 		// print_data(buff, size);

// 		// for(unsigned long long size_k = size; size_k >= 2; size_k /= 2) {
// 		for(unsigned long long size_k_p = size_p; size_k_p >= 1; size_k_p--) {
// 			int size_k = 1<<size_k_p;
// 			// printf("Before i = %d, size_k = %d\n", i, size_k);
// 			// print_data(buff, size);
// 			for(unsigned long long j = threadIdx.x; j < (size >> 1); j += blockDim.x) {
// 				unsigned long long z = (j >> (size_k_p-1)) * size_k + (j & ((1 << (size_k_p-1)) - 1));
// 				if ((buff[z] > buff[z + (size_k >> 1)]) != (((i+z) / sign_shift) & 1)) {
// 					swap_GPU(buff[z], buff[z + (size_k >> 1)]);
// 				}
// 			}
// 			// printf(" After i = %d, size_k = %d\n", i, size_k);
// 			// print_data(buff, size);
// 			__syncthreads();
// 		}
// 		for(int k = threadIdx.x; k < size; k += blockDim.x)
// 			data[i + k] = buff[k];
// 		__syncthreads();
// 	}
// }





__global__ void  B_shared_kernal(int* data, int size_p, int N, int sign_shift_p) {
	int sign_shift = 1<< sign_shift_p;
	// int size = 1<<size_p;
	__shared__ int buff[512];

	// int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// int offset = blockDim.x * gridDim.x;

	// for(int i = idx; i < N; i += offset) {
		



	for(unsigned long long i = (unsigned long long)blockIdx.x * 512; i < N; i += (unsigned long long)gridDim.x * 512) {
		for(int k = threadIdx.x; k < 512; k += blockDim.x)
			buff[k] = data[i + k];
		__syncthreads();
		// printf("i = %d\n", i);
		// print_data(buff, size);

		// for(unsigned long long size_k = size; size_k >= 2; size_k /= 2) {
		for(unsigned long long size_k_p = size_p; size_k_p >= 1; size_k_p--) {
			unsigned long long size_k = 1<<size_k_p;
			for(unsigned long long j = threadIdx.x; j < 256; j += blockDim.x) {
			// for(unsigned long long j = threadIdx.x; j < (size >> 1); j += blockDim.x) {
				unsigned long long z = (unsigned long long)(j >> (size_k_p-1)) * size_k + (j & ((1 << (size_k_p-1)) - 1));
				if ((buff[z] > buff[z + (size_k >> 1)]) != (((i+z) / sign_shift) & 1)) {
					swap_GPU(buff[z], buff[z + (size_k >> 1)]);
				}
			}
			// printf(" After i = %d, size_k = %d\n", i, size_k);
			// print_data(buff, size);
			__syncthreads();
		}
		for(int k = threadIdx.x; k < 512; k += blockDim.x)
			data[i + k] = buff[k];
		__syncthreads();
	}
}


__global__ void  B_global_kernal(int* data, int size_p, int N, int sign_shift_p) {
	unsigned long long sign_shift = 1<< sign_shift_p;
	unsigned long long size = 1<<size_p;
	for(unsigned long long i = (unsigned long long)blockIdx.x * size; i < N; i += (unsigned long long)gridDim.x * size) {
		for(unsigned long long j = threadIdx.x; j < size / 2; j += blockDim.x) {
			if ((data[i+j] > data[i+j + size / 2]) == (((i+j) / sign_shift) % 2 == 0)) {
				swap_GPU(data[i+j], data[i+j + size / 2]);
			}
		}
	}
}

void M(int* data, int size, int N) {
	for(int i = size; i >= 2; i /= 2) {
		B(data, i, N, size);

	}
}

void M_GPU(int* dev_data, int size, int N) {
	for(int i = size; i >= 1; i--) {
		if(i <= 9) {
			B_shared_kernal<<<128, 128>>>(dev_data, i, N, size);
			break;
		}
		B_global_kernal<<<128, 1024>>>(dev_data, i, N, size);

		CSC(hipGetLastError());
		// print_data(data, N);
	}
}

unsigned long upper_power_of_two(unsigned long v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;

}

int main() {
	unsigned int  n;


	// scanf("%d", &n);
	fread(&n, sizeof(int), 1, stdin);
	unsigned int  N = upper_power_of_two(n);
	
	int *data = (int *)malloc(sizeof(int) * N);

	fread(data, sizeof(int), n, stdin);
	// for(int i = 0; i < n; i++) {
	// 	scanf("%d", &data[i]);
	// }

	for(int i = n; i < N; i++) {
		data[i] = INT_MAX;
	}

	

	
	int *dev_data;
	CSC(hipMalloc(&dev_data, sizeof(int) * N));
	CSC(hipMemcpy(dev_data, data, sizeof(int) * N, hipMemcpyHostToDevice));

	// struct timeval start, stop;
	// double secs = 0;
	// gettimeofday(&start, NULL);

	// for(int i = 2; i <= N; i*=2) {
	// 	M(data, i, N);
	// }

	for(int i = 1; (1<<i) <= N; i++) {
		M_GPU(dev_data, i, N);
	}
	CSC(hipGetLastError());

	CSC(hipMemcpy(data, dev_data, sizeof(int) * N, hipMemcpyDeviceToHost));
	CSC(hipFree(dev_data));


	// gettimeofday(&stop, NULL);
	// secs = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);
	// printf("time taken %f\n",secs);


	// for(int i = 0; i < N; i++) {
	// 	printf("%d ", data[i]);
	// }
	// printf("\n");
	fwrite(data, sizeof(int), n, stdout);

	free(data);

	return 0;
}
