#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <math.h>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

using namespace std;

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

<<<<<<< Updated upstream
__global__ void prewitt(uchar4 * res, int w, int h) {
    for (int y = blockDim.y * blockIdx.y + threadIdx.y; y < h; y += blockDim.y * gridDim.y) {
		for (int x = blockDim.x * blockIdx.x + threadIdx.x; x < w; x += blockDim.x * gridDim.x) {
			uchar4 p = tex2D(tex, x, y);
            uchar4 piv;
            float gx = 0;
            float gy = 0;
            
            int sx1[2] = {max(min(x+1, w-1),0),max(min(x-1, w-1),0)};
            int sy1[3] = {max(min(y, h-1),0),max(min(y+1, h-1),0),max(min(y-1, h-1),0)};
            
			for (int j=0;j<3;j++) {
			    piv = tex2D(tex, sx1[0], sy1[j]);
			    gx += (float)piv.x * 0.299 + (float)piv.y * 0.587 + (float)piv.z * 0.114;
			    piv = tex2D(tex, sx1[1], sy1[j]);
			    gx -= (float)piv.x * 0.299 + (float)piv.y * 0.587 + (float)piv.z * 0.114;
			}
            
            int sx2[3] = {max(min(x, w-1),0),max(min(x-1, w-1),0),max(min(x+1, w-1),0)};
            int sy2[2] = {max(min(y+1, h-1),0),max(min(y-1, h-1),0)};
            
            for (int j=0;j<3;j++) {
                piv = tex2D(tex, sx2[j], sy2[0]);
                gy += (float)piv.x * 0.299 + (float)piv.y * 0.587 + (float)piv.z * 0.114;
                piv = tex2D(tex, sx2[j], sy2[1]);
                gy -= (float)piv.x * 0.299 + (float)piv.y * 0.587 + (float)piv.z * 0.114;
            } 
            
			float g = sqrt(gx*gx + gy*gy);      
			unsigned char mean = (unsigned char) min(255,(int)g);
			res[y * w + x] = make_uchar4(mean, mean, mean, p.w);
		}
=======
__global__ void prewitt (int x, int y, Pixel * pic, Pixel * res) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < x * y; i += gridDim.x * blockDim.x) {
        int yt = i / x;
        int xt = i - yt * x;
        float gx = 0;
        float gy = 0;

        int sx1[2] = {max(min(xt+1, x-1),0),max(min(xt-1, x-1),0)};
        int sy1[3] = {max(min(yt, y-1),0),max(min(yt+1, y-1),0),max(min(yt-1, y-1),0)};

        for (int j=0;j<3;j++) gx += (float)pic[sx1[0]+sy1[j]*x].r;
        for (int j=0;j<3;j++) gx -= (float)pic[sx1[1]+sy1[j]*x].r;
        
        int sx2[3] = {max(min(xt, x-1),0),max(min(xt-1, x-1),0),max(min(xt+1, x-1),0)};
        int sy2[2] = {max(min(yt+1, y-1),0),max(min(yt-1, y-1),0)};

        for (int j=0;j<3;j++) gy += (float)pic[sx2[j]+sy2[0]*x].r;
        for (int j=0;j<3;j++) gy -= (float)pic[sx2[j]+sy2[1]*x].r;

        float g = min(255.0,sqrt(gx*gx + gy*gy));

        res[i].r = (unsigned char)g;
        res[i].g = (unsigned char)g;
        res[i].b = (unsigned char)g;
        res[i].a = pic[i].a;
>>>>>>> Stashed changes
    }
}

int main() {
    string filename1, filename2;
    cin >> filename1 >> filename2;
<<<<<<< Updated upstream
	int w, h;
	
	FILE * f = fopen(filename1.c_str(), "rb");
	fread(&w, sizeof(int), 1, f);
	fread(&h, sizeof(int), 1, f);
	uchar4 * data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, f);
	fclose(f);
=======

    Image pic;
    pic.load(string(filename1));
    pic.ink();

    Pixel *dev_pic, *dev_res;
    hipMalloc(&dev_pic, sizeof(Pixel) * pic.x * pic.y);
    hipMalloc(&dev_res, sizeof(Pixel) * pic.x * pic.y);
>>>>>>> Stashed changes

	// Подготовка данных для текстуры
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));
	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));
	// Подготовка текстурной ссылки, настройка интерфейса работы с данными
	tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;		// Без интерполяции при обращении по дробным координатам
	tex.normalized = false;						// Режим нормализации координат: без нормализации

<<<<<<< Updated upstream
	// Связываем интерфейс с данными
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 * dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));
	
	// Prepare
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
// Start record
hipEventRecord(start, 0);
// Do something on GPU

	prewitt<<<dim3(16, 16), dim3(16, 32)>>>(dev_out, w, h);
	
// Stop event
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
// Clean up:
hipEventDestroy(start);
hipEventDestroy(stop);

	CSC(hipGetLastError());

	CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	// Отвязываем данные от текстурной ссылки
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	f = fopen(filename2.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, f);
	fwrite(&h, sizeof(int), 1, f);
	fwrite(data, sizeof(uchar4), w * h, f);
	fclose(f);

	free(data);
	return 0;
=======
    prewitt <<<2,2>>>(pic.x, pic.y, dev_pic, dev_res);

    hipDeviceSynchronize();

    hipMemcpy(pic.pixels, dev_res, sizeof(Pixel) * pic.x * pic.y, hipMemcpyDeviceToHost);

    hipFree(dev_pic);
    hipFree(dev_res);

    pic.save(string(filename2));
    return 0;
>>>>>>> Stashed changes
}
