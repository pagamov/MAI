#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <math.h>
#include <mpi.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

using namespace std;

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)


#define _i(i, j, k) (((k) + 1) * (nx + 2) * (ny + 2) + ((j) + 1) * (nx + 2) + (i) + 1)
#define _iD(i, j, k) ((k + 1) * (dim[0] + 2) * (dim[1] + 2) + (j + 1) * (dim[0] + 2) + i + 1)
#define _ib(i, j, k) ((k) * box[0] * box[1] + (j) * box[0] + (i))

__global__ void kernel_init(double* data, int nx, int ny, int nz, double u0) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < nz; k+=blockDim.z * gridDim.z)
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j+=blockDim.y * gridDim.y)
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    			data[_i(i, j, k)] = u0;
}

__global__ void kernel_LR(double* data, int nx, int ny, int nz, double bc, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j+=blockDim.x * gridDim.x)
    		data[_i(x_ind, j, k)] = bc;
}

__global__ void kernel_FB(double* data, int nx, int ny, int nz, double bc, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		data[_i(i, y_ind, k)] = bc;
}

__global__ void kernel_DU(double* data, int nx, int ny, double bc, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		data[_i(i, j, z_ind)] = bc;
}

__global__ void kernel_send_LR(double* buf, double* data, int nx, int ny, int nz, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j+=blockDim.x * gridDim.x)
    		buf[k * ny + j] = data[_i(x_ind, j, k)];
}

__global__ void kernel_recive_LR(double* buf, double* data, int nx, int ny, int nz, int x_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int j = blockDim.x * blockIdx.x + threadIdx.x; j < ny; j+=blockDim.x * gridDim.x)
    		data[_i(x_ind, j, k)] = buf[k * ny + j];
}

__global__ void kernel_send_FB(double* buf, double* data, int nx, int ny, int nz, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		buf[k * nx + i] = data[_i(i, y_ind, k)];
}

__global__ void kernel_recive_FB(double* buf, double* data, int nx, int ny, int nz, int y_ind) {
    for (int k = blockDim.y * blockIdx.y + threadIdx.y; k < nz; k+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		data[_i(i, y_ind, k)] = buf[k * nx + i];
}

__global__ void kernel_send_DU(double* buf, double* data, int nx, int ny, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		buf[j * nx + i] = data[_i(i, j, z_ind)];
}

__global__ void kernel_recive_DU(double* buf, double* data, int nx, int ny, int z_ind) {
    for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j+=blockDim.y * gridDim.y)
    	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x)
    		data[_i(i, j, z_ind)] = buf[j * nx + i];
}

__global__ void kernel_get_vals(double* data, double* next, int nx, int ny, int nz, double hx, double hy, double hz) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < nz; k+=blockDim.z * gridDim.z) {
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < ny; j+=blockDim.y * gridDim.y) {
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nx; i+=blockDim.x * gridDim.x) {
    			next[_i(i, j, k)] = 0.5 * ((data[_i(i + 1, j, k)] + data[_i(i - 1, j, k)]) / (hx * hx) +
						(data[_i(i, j + 1, k)] + data[_i(i, j - 1, k)]) / (hy * hy) +
						(data[_i(i, j, k + 1)] + data[_i(i, j, k - 1)]) / (hz * hz)) /
						(1.0 / (hx * hx) + 1.0 / (hy * hy) + 1.0 / (hz * hz));
    		}
    	}
    }
}

__global__ void kernel_get_diffs(double* data, double* next, int nx, int ny, int nz) {
    for (int k = blockDim.z * blockIdx.z + threadIdx.z - 1; k <= nz; k+=blockDim.z * gridDim.z)
    	for (int j = blockDim.y * blockIdx.y + threadIdx.y - 1; j <= ny; j+=blockDim.y * gridDim.y)
    		for (int i = blockDim.x * blockIdx.x + threadIdx.x - 1; i <= nx; i+=blockDim.x * gridDim.x)
    			data[_i(i, j, k)] = ((i != -1) && (j != -1) && (k != -1) && (i != nx) && (j != ny) && (k != nz)) * fabs(next[_i(i, j, k)] - data[_i(i, j, k)]);
}


int main(int argc, char * argv[]) {
	int ib, jb, kb;
	int id, numproc;
	double lx, ly, lz, hx, hy, hz, down, up, left, right, front, back, u_0;
	double eps, cur_eps;
	double *temp, *buff;
	char filename[100];
    int box[3];
    int dim[3];
    bool f = true;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	int deviceCount;
	CSC(hipGetDeviceCount(&deviceCount));
	CSC(hipSetDevice(id % deviceCount));

	if (id == 0) {
        cin >> box[0] >> box[1] >> box[2];
        cin >> dim[0] >> dim[1] >> dim[2];
		scanf("%s", filename);
        cin >> eps;
        cin >> lx >> ly >> lz;
        cin >> down >> up >> left >> right >> front >> back >> u_0;
	}

    MPI_Bcast(box, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(dim, 3, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lz, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&down, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&up, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&left, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&right, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&front, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&back, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&u_0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(filename, 100, MPI_CHAR, 0, MPI_COMM_WORLD);

	kb = id / (box[0] * box[1]);
	jb = id % (box[0] * box[1]) / box[0];
	ib = id % (box[0] * box[1]) % box[0];

	hx = lx / (dim[0] * box[0]);
	hy = ly / (dim[1] * box[1]);
	hz = lz / (dim[2] * box[2]);

	double *dev_data;
	double *dev_next;
	CSC(hipMalloc(&dev_data, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2)));
	CSC(hipMalloc(&dev_next, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2)));

	buff = (double*)malloc(sizeof(double) * max(dim[0], dim[1]) * max(dim[1], dim[2]));
	int buffer_size;
	MPI_Pack_size(max(dim[0], dim[1]) * max(dim[1], dim[2]), MPI_DOUBLE, MPI_COMM_WORLD, &buffer_size);
	buffer_size = 6 * (buffer_size + MPI_BSEND_OVERHEAD);
	double* buffer = (double*)malloc(buffer_size);
	MPI_Buffer_attach(buffer, buffer_size);
	double* dev_buff;
	CSC(hipMalloc(&dev_buff, sizeof(double) * max(dim[0], dim[1]) * max(dim[1], dim[2])));
    double * allgbuff = (double *)malloc(sizeof(double) * box[0] * box[1] * box[2]);
	kernel_init<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dim[0], dim[1], dim[2], u_0);
	CSC(hipGetLastError());

	while (f) {
		MPI_Barrier(MPI_COMM_WORLD);

		if (ib + 1 < box[0]) {
			kernel_send_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[0]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[1] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib + 1, jb, kb), id, MPI_COMM_WORLD);
		}

		if (jb + 1 < box[1]) {
			kernel_send_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[1]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb + 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb + 1 < box[2]) {
			kernel_send_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2]-1);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[1], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb + 1), id, MPI_COMM_WORLD);
		}

		if (ib > 0) {
			kernel_send_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[1] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib - 1, jb, kb), id, MPI_COMM_WORLD);
		}

		if (jb > 0) {
			kernel_send_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[2], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb - 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb > 0) {
			kernel_send_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], 0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(buff, dev_buff, sizeof(double) * dim[0] * dim[1], hipMemcpyDeviceToHost));
			MPI_Bsend(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb - 1), id, MPI_COMM_WORLD);
		}

		if (ib > 0) {
			MPI_Recv(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib - 1, jb, kb), _ib(ib - 1, jb, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[1] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], -1);
			CSC(hipGetLastError());
		} else {
			kernel_LR<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], left, -1);
			CSC(hipGetLastError());
		}

		if (jb > 0) {
			MPI_Recv(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb - 1, kb), _ib(ib, jb - 1, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], -1);
			CSC(hipGetLastError());
		} else {
			kernel_FB<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], front, -1);
			CSC(hipGetLastError());
		}

		if (kb > 0) {
			MPI_Recv(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb - 1), _ib(ib, jb, kb - 1), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[1], hipMemcpyHostToDevice));
			kernel_recive_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], -1);
			CSC(hipGetLastError());
		} else {
			kernel_DU<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], down, -1);
			CSC(hipGetLastError());
		}

		if (ib + 1 < box[0]) {
			MPI_Recv(buff, dim[1] * dim[2], MPI_DOUBLE, _ib(ib + 1, jb, kb), _ib(ib + 1, jb, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[1] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_LR<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[0]);
			CSC(hipGetLastError());
		} else {
			kernel_LR<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], right, dim[0]);
			CSC(hipGetLastError());
		}

		if (jb + 1 < box[1]) {
			MPI_Recv(buff, dim[0] * dim[2], MPI_DOUBLE, _ib(ib, jb + 1, kb), _ib(ib, jb + 1, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[2], hipMemcpyHostToDevice));
			kernel_recive_FB<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2], dim[1]);
			CSC(hipGetLastError());
		} else {
			kernel_FB<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], dim[2], back, dim[1]);
			CSC(hipGetLastError());
		}

		if (kb + 1 < box[2]) {
			MPI_Recv(buff, dim[0] * dim[1], MPI_DOUBLE, _ib(ib, jb, kb + 1), _ib(ib, jb, kb + 1), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(dev_buff, buff, sizeof(double) * dim[0] * dim[1], hipMemcpyHostToDevice));
			kernel_recive_DU<<<dim3(32,32), dim3(32,32)>>>(dev_buff, dev_data, dim[0], dim[1], dim[2]);
			CSC(hipGetLastError());
		} else {
			kernel_DU<<<dim3(32,32), dim3(32,32)>>>(dev_data, dim[0], dim[1], up, dim[2]);
			CSC(hipGetLastError());
		}

		MPI_Barrier(MPI_COMM_WORLD);
		cur_eps = 0.0;

		kernel_get_vals<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dev_next, dim[0], dim[1], dim[2], hx, hy, hz);
		CSC(hipGetLastError());

		kernel_get_diffs<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(dev_data, dev_next, dim[0], dim[1], dim[2]);
		CSC(hipGetLastError());

		thrust::device_ptr<double> diffs = thrust::device_pointer_cast(dev_data);
        thrust::device_ptr<double> max_eps = thrust::max_element(diffs, diffs + (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2));
        cur_eps = *max_eps;

        MPI_Allgather(&cur_eps, 1, MPI_DOUBLE, allgbuff, 1, MPI_DOUBLE, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
        f = false;
        for (int i = 0; i < box[0] * box[1] * box[2]; i++)
            if (allgbuff[i] > eps)
                f = true;

		temp = dev_next;
		dev_next = dev_data;
		dev_data = temp;
	}

	double * data = (double *)malloc(sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2));
	hipMemcpy(data, dev_data, sizeof(double) * (dim[0] + 2) * (dim[1] + 2) * (dim[2] + 2), hipMemcpyDeviceToHost);

	char * bf = (char*)malloc(sizeof(char) * dim[0] * dim[1] * dim[2] * 14);
	memset(bf, ' ', sizeof(char) * dim[0] * dim[1] * dim[2] * 14);

	for (int k = 0; k < dim[2]; k++)
		for (int j = 0; j < dim[1]; j++)
			for (int i = 0; i < dim[0]; i++)
				sprintf(bf + (k * dim[0] * dim[1] + j * dim[0] + i) * 14, "%.6e", data[_iD(i, j, k)]);

	for (int i = 0; i < dim[0] * dim[1] * dim[2] * 14; i++)
		if (bf[i] == '\0')
			bf[i] = ' ';

	MPI_File fp;
	MPI_Datatype filetype;
	int sizes[] = {dim[2] * box[2], dim[1] * box[1], dim[0] * box[0] * 14};
	int subsizes[] = {dim[2], dim[1], dim[0] * 14};
	int starts[] = {id / (box[0] * box[1]) * dim[2], id % (box[0] * box[1]) / box[0] * dim[1],  id % (box[0] * box[1]) % box[0] * dim[0] * 14};
	MPI_Type_create_subarray(3, sizes, subsizes, starts, MPI_ORDER_C, MPI_CHAR, &filetype);
	MPI_Type_commit(&filetype);

	MPI_File_delete(filename, MPI_INFO_NULL);
	MPI_File_open(MPI_COMM_WORLD, filename, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);
	MPI_File_set_view(fp, 0, MPI_CHAR, filetype, "native", MPI_INFO_NULL);
	MPI_File_write_all(fp, bf, dim[0] * dim[1] * dim[2] * 14, MPI_CHAR, MPI_STATUS_IGNORE);
	MPI_File_close(&fp);

	MPI_Type_free(&filetype);
	MPI_Finalize();
    CSC(hipFree(dev_data));
	CSC(hipFree(dev_next));
	CSC(hipFree(dev_buff));
	free(data);
	free(buff);
	free(bf);
	return 0;
}
