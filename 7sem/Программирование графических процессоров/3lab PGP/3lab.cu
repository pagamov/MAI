
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <string.h>
#include <math.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

typedef struct { double x, y, z; } LL3; 									    // +4 294 967 295 ~ 255 * 4100 * 4100
typedef struct { int w, h, n; } info;											// pic 4100 by 4100 of one claster max pixel
																				// mb need to be changed to higher
typedef struct { double x, y, z; } D3;
__constant__ D3 CLASSES[32];													// contant classes no more than 32 by default
__constant__ info inf[1];														// some else param that never changes

#define SIZE_OF_PIC sizeof(uchar4) * w * h
#define SIZE_OF_CL sizeof(D3) * n
#define SIZE_OF_CLUINT sizeof(LL3) * n
#define SIZE_OF_INFO sizeof(info)

// texture<uchar4, 2, cudaReadModeElementType> tex;

__global__ void Kmean(uchar4 * pic) {
    for (int y = blockDim.y * blockIdx.y + threadIdx.y; y < inf[0].h; y += blockDim.y * gridDim.y) {
        for (int x = blockDim.x * blockIdx.x + threadIdx.x; x < inf[0].w; x += blockDim.x * gridDim.x) {
            double maxDist = sqrt((double)3*(255*255))+(double)1;
			// uchar4 piv = tex2D(tex, x, y);
			uchar4 piv = pic[x + inf[0].w * y];
            for (int i = 0; i < inf[0].n; i++) {
                double pivDist = sqrt( 					   						      \
					(((double)piv.x-CLASSES[i].x) * ((double)piv.x-CLASSES[i].x)) + \
					(((double)piv.y-CLASSES[i].y) * ((double)piv.y-CLASSES[i].y)) + \
					(((double)piv.z-CLASSES[i].z) * ((double)piv.z-CLASSES[i].z))   \
				);
                if (pivDist < maxDist) {
					pic[x + y * inf[0].w].w = (unsigned char)i;
                    maxDist = pivDist;
                }
            }
        }
    }
}

int main() {
	double eps = 0.1;
    string filename1, filename2;
    int w, h, n, x, y, flag = 1;
    cin >> filename1 >> filename2 >> n;
	
	FILE * f = fopen(filename1.c_str(), "rb");									// read data section
	fread(&w, sizeof(int), 1, f);
	fread(&h, sizeof(int), 1, f);
	uchar4 * data = (uchar4 *)malloc(SIZE_OF_PIC); 								// malloc data
	fread(data, sizeof(uchar4), w * h, f);
	fclose(f);
	
	uchar4 * DEV_DATA;															// make dev struct for kernel
	CSC(hipMalloc(&DEV_DATA, SIZE_OF_PIC));									// cuda malloc dev_pic
	CSC(hipMemcpy(DEV_DATA, data, SIZE_OF_PIC, hipMemcpyHostToDevice));
	
	info infHost[1];															// infHost
	infHost[0].w = w; infHost[0].h = h; infHost[0].n = n;						// copy info data and never touch again
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(inf), infHost, SIZE_OF_INFO, 0, hipMemcpyHostToDevice));
	
	D3 clHost[32];															// read data for classes
    for (int i = 0; i < n; i++) {
		cin >> x >> y;
		clHost[i].x = (double)data[x + w * y].x;
		clHost[i].y = (double)data[x + w * y].y;
		clHost[i].z = (double)data[x + w * y].z;
	} 																			// we ll touch it every cicle
	
	LL3 * CLASSES_NEW = 	(LL3 *)malloc(SIZE_OF_CLUINT);						// malloc CLASSES_NEW
	int * CLASSES_NEW_NUM = (int *)malloc(sizeof(int) * n);						// test
	
	// // Подготовка данных для текстуры
	// cudaArray *arr;
	// cudaChannelFormatDesc ch = cudaCreateChannelDesc<uchar4>();
	// CSC(cudaMallocArray(&arr, &ch, w, h));
	// CSC(cudaMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, cudaMemcpyHostToDevice));
	// // Подготовка текстурной ссылки, настройка интерфейса работы с данными
	// tex.addressMode[0] = cudaAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
	// tex.addressMode[1] = cudaAddressModeClamp;
	// tex.channelDesc = ch;
	// tex.filterMode = cudaFilterModePoint;		// Без интерполяции при обращении по дробным координатам
	// tex.normalized = false;						// Режим нормализации координат: без нормализации
	// 
	// // Связываем интерфейс с данными
	// CSC(cudaBindTextureToArray(tex, arr, ch));
	// 
	while (flag) {
		CSC(hipMemcpyToSymbol(HIP_SYMBOL(CLASSES), clHost, SIZE_OF_CL, 0, hipMemcpyHostToDevice));
		Kmean <<<dim3(16, 16), dim3(32, 32)>>> (DEV_DATA);						// find new clasters
		CSC(hipMemcpy(data, DEV_DATA, SIZE_OF_PIC, hipMemcpyDeviceToHost));
		
		for (int i = 0; i < n; i++) {
			CLASSES_NEW_NUM[i] = 0;
			CLASSES_NEW[i].x = 0;
			CLASSES_NEW[i].y = 0;
			CLASSES_NEW[i].z = 0;
		}
		
		for (int y = 0; y < h; y++) {
			for (int x = 0; x < w; x++) {
				int idx = data[y * w + x].w;
				CLASSES_NEW[idx].x += data[y * w + x].x;
				CLASSES_NEW[idx].y += data[y * w + x].y;
				CLASSES_NEW[idx].z += data[y * w + x].z;
				CLASSES_NEW_NUM[idx] += 1;
			}
		}
	
		for (int i = 0; i < n; i++) {
			if (CLASSES_NEW_NUM[i] != 0) {
				CLASSES_NEW[i].x /= CLASSES_NEW_NUM[i];
				CLASSES_NEW[i].y /= CLASSES_NEW_NUM[i];
				CLASSES_NEW[i].z /= CLASSES_NEW_NUM[i];
			}
		}
		
		flag = 0;
		for (int i = 0; i < n; i++) {
			if (!(											\
				(abs(clHost[i].x - CLASSES_NEW[i].x) < eps) && \
				(abs(clHost[i].y - CLASSES_NEW[i].y) < eps) && \
				(abs(clHost[i].z - CLASSES_NEW[i].z) < eps)	\
				)) {
					flag = 1;
			}
		}
		
		for (int i = 0; i < n; i++) {
			clHost[i].x = CLASSES_NEW[i].x;
			clHost[i].y = CLASSES_NEW[i].y;
			clHost[i].z = CLASSES_NEW[i].z;
		}
	}
	// 
	// // Отвязываем данные от текстурной ссылки
	// CSC(cudaUnbindTexture(tex));
	// CSC(cudaFreeArray(arr));
	// 
	CSC(hipFree(DEV_DATA));														// cuda free dev_pic

	f = fopen(filename2.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, f);
	fwrite(&h, sizeof(int), 1, f);
	fwrite(data, sizeof(uchar4), w * h, f);
	fclose(f);

	free(data);																	// free data
	free(CLASSES_NEW_NUM);
	free(CLASSES_NEW);															// free CLASSES_NEW
	return 0;
}
