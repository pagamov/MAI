
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(double *arr, int n) {
	int i, idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for(i = idx; i < n; i += offset)	
		arr[i] = arr[i] < 0 ? -arr[i] : arr[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main() {
	int i, n;
	scanf("%d",&n);
	double *arr = (double *)malloc(sizeof(double) * n);
	for(i = 0; i < n; i++)
		scanf("%lf", &arr[i]);

	double *dev_arr;
	gpuErrchk(hipMalloc(&dev_arr, sizeof(double) * n));
	gpuErrchk(hipMemcpy(dev_arr, arr, sizeof(double) * n, hipMemcpyHostToDevice));

	kernel<<<256, 256>>>(dev_arr, n);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMemcpy(arr, dev_arr, sizeof(double) * n, hipMemcpyDeviceToHost));
	gpuErrchk( hipFree(dev_arr));
	for(i = 0; i < n; i++)
		printf("%lf ", arr[i]);
	printf("\n");
	free(arr);
	return 0;
}
