#include "hip/hip_runtime.h"
// nvcc opengl_2.cu -lGL -lGLU -lglut -lGLEW
// ./a.out

// Для linux нужно поставить пакеты: libgl1-mesa-dev libglew-dev freeglut3-dev
// sudo apt-get install libgl1-mesa-dev libglew-dev freeglut3-dev

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

typedef unsigned char uchar;

#define sqr3(x) ((x)*(x)*(x))
#define sqr(x) ((x)*(x))

struct t_item {
	float x;
	float y;
	float z;
	float dx;
	float dy;
	float dz;
	float q;
};

t_item item;

int w = 1024, h = 648;

float x = -1.5, y = -1.5, z = 1.0;
float dx = 0.0, dy = 0.0, dz = 0.0;
float yaw = 0.0, pitch = 0.0;
float dyaw = 0.0, dpitch = 0.0;

float speed = 0.05;

const float a2 = 15.0;			
const int np = 100;				// Размер текстуры пола

GLUquadric* quadratic;			// quadric объекты - это геометрические фигуры 2-го порядка, т.е. сфера, цилиндр, диск, конус. 

hipGraphicsResource *res;		
GLuint textures[2];				// Массив из текстурных номеров
GLuint vbo;						// Номер буфера


__global__ void kernel(uchar4 *data, t_item item, float t) {	// Генерация текстуры пола на GPU
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int i, j;
	float x, y, fg, fb;
	for(i = idx; i < np; i += offsetx)
		for(j = idy; j < np; j += offsety) {
			x = (2.0 * i / (np - 1.0) - 1.0) * a2;
			y = (2.0 * j / (np - 1.0) - 1.0) * a2;
			fb = 100.0 * (sin(0.1 * x * x + t) + cos(0.1 * y * y + t * 0.6) + sin(0.1 * x * x + 0.1 * y * y + t * 0.3));
			fg = 10000.0 * item.q / (sqr(x - item.x) + sqr(y - item.y) + sqr(item.z) + 0.001);
			fg = min(max(0.0f, fg), 255.0f);
			fb = min(max(0.0f, fb), 255.0f);
			data[j * np + i] = make_uchar4(0, (int)fg, (int)fb, 255);
		}
}

void display() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	// Задаем "объектив камеры"
	gluPerspective(90.0f, (GLfloat)w/(GLfloat)h, 0.1f, 100.0f);


	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	// Задаем позицию и направление камеры
	gluLookAt(x, y, z,
				x + cos(yaw) * cos(pitch),
				y + sin(yaw) * cos(pitch),
				z + sin(pitch),
				0.0f, 0.0f, 1.0f);

	glBindTexture(GL_TEXTURE_2D, textures[0]);	// Задаем текстуру


	static float angle = 0.0;
	glPushMatrix();
		glTranslatef(item.x, item.y, item.z);	// Задаем координаты центра сферы
		glRotatef(angle, 0.0, 0.0, 1.0);
		gluSphere(quadratic, 2.5f, 32, 32);
	glPopMatrix();
	angle += 0.15;


	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);	// Делаем активным буфер с номером vbo
	glBindTexture(GL_TEXTURE_2D, textures[1]);	// Делаем активной вторую текстуру
	glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)np, (GLsizei)np, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL); 
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);	// Деактивируем буфер
	// Последний параметр NULL в glTexImage2D говорит о том что данные для текстуры нужно брать из активного буфера
	
	glBegin(GL_QUADS);			// Рисуем пол
		glTexCoord2f(0.0, 0.0);
		glVertex3f(-a2, -a2, 0.0);

		glTexCoord2f(1.0, 0.0);
		glVertex3f(a2, -a2, 0.0);

		glTexCoord2f(1.0, 1.0);
		glVertex3f(a2, a2, 0.0);

		glTexCoord2f(0.0, 1.0);
		glVertex3f(-a2, a2, 0.0);
	glEnd();

	
	glBindTexture(GL_TEXTURE_2D, 0);			// Деактивируем текстуру

	// Отрисовка каркаса куба				
	glLineWidth(2);								// Толщина линий				
	glColor3f(0.5f, 0.5f, 0.5f);				// Цвет линий
	glBegin(GL_LINES);							// Последующие пары вершин будут задавать линии
		glVertex3f(-a2, -a2, 0.0);
		glVertex3f(-a2, -a2, 2.0 * a2);

		glVertex3f(a2, -a2, 0.0);
		glVertex3f(a2, -a2, 2.0 * a2);

		glVertex3f(a2, a2, 0.0);
		glVertex3f(a2, a2, 2.0 * a2);

		glVertex3f(-a2, a2, 0.0);
		glVertex3f(-a2, a2, 2.0 * a2);
	glEnd();

	glBegin(GL_LINE_LOOP);						// Все последующие точки будут соеденены замкнутой линией
		glVertex3f(-a2, -a2, 0.0);
		glVertex3f(a2, -a2, 0.0);
		glVertex3f(a2, a2, 0.0);
		glVertex3f(-a2, a2, 0.0);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex3f(-a2, -a2, 2.0 * a2);
		glVertex3f(a2, -a2, 2.0 * a2);
		glVertex3f(a2, a2, 2.0 * a2);
		glVertex3f(-a2, a2, 2.0 * a2);
	glEnd();

	glColor3f(1.0f, 1.0f, 1.0f);

	glutSwapBuffers();
}



void update() {
//	dz -= 0.0001;			// Гравитация

	float v = sqrt(dx * dx + dy * dy + dz * dz);
	if (v > speed) {		// Ограничение максимальной скорости
		dx *= speed / v;
		dy *= speed / v;
		dz *= speed / v;
	}
	x += dx; dx *= 0.99;
	y += dy; dy *= 0.99;
	z += dz; dz *= 0.99;
	if (z < 1.0) {			// Пол, ниже которого камера не может переместиться
		z = 1.0;
		dz = 0.0;
	}
	if (fabs(dpitch) + fabs(dyaw) > 0.0001) {	// Вращение камеры
		yaw += dyaw;
		pitch += dpitch;
		pitch = min(M_PI / 2.0 - 0.0001, max(-M_PI / 2.0 + 0.0001, pitch));
		dyaw = dpitch = 0.0;
	}

	float w = 0.9999, e0 = 1e-3, dt = 0.01, K = 50.0;

// Замедление
	item.dx *= w;
	item.dy *= w;
	item.dz *= w;

// Отталкивание от стен
	item.dx += item.q * item.q * K * (item.x - a2) / (sqr3(fabs(item.x - a2)) + e0) * dt;
	item.dx += item.q * item.q * K * (item.x + a2) / (sqr3(fabs(item.x + a2)) + e0) * dt;

	item.dy += item.q * item.q * K * (item.y - a2) / (sqr3(fabs(item.y - a2)) + e0) * dt;
	item.dy += item.q * item.q * K * (item.y + a2) / (sqr3(fabs(item.y + a2)) + e0) * dt;

	item.dz += item.q * item.q * K * (item.z - 2 * a2) / (sqr3(fabs(item.z - 2 * a2)) + e0) * dt;
	item.dz += item.q * item.q * K * (item.z + 0.0) / (sqr3(fabs(item.z + 0.0)) + e0) * dt;

// Отталкивание от камеры
	float l = sqrt(sqr(item.x - x) + sqr(item.y - y) + sqr(item.z - z));
	item.dx += 3.0 * item.q * K * (item.x - x) / (l * l * l + e0) * dt;
	item.dy += 3.0 * item.q * K * (item.y - y) / (l * l * l + e0) * dt;
	item.dz += 3.0 * item.q * K * (item.z - z) / (l * l * l + e0) * dt;

	item.x += item.dx * dt;
	item.y += item.dy * dt;
	item.z += item.dz * dt;

	static float t = 0.0;
	uchar4* dev_data;
	size_t size;
	hipGraphicsMapResources(1, &res, 0);		// Делаем буфер доступным для CUDA
	hipGraphicsResourceGetMappedPointer((void**) &dev_data, &size, res);	// Получаем указатель на память буфера
	kernel<<<dim3(32, 32), dim3(32, 8)>>>(dev_data, item, t);		
	hipGraphicsUnmapResources(1, &res, 0);		// Возращаем буфер OpenGL'ю что бы он мог его использовать
	t += 0.01;

	glutPostRedisplay();	// Перерисовка
}

void keys(unsigned char key, int x, int y) {	// Обработка кнопок
	switch (key) {
		case 'w':                 // "W" Движение вперед
			dx += cos(yaw) * cos(pitch) * speed;
			dy += sin(yaw) * cos(pitch) * speed;
			dz += sin(pitch) * speed;
		break;
		case 's':                 // "S" Назад
			dx += -cos(yaw) * cos(pitch) * speed;
			dy += -sin(yaw) * cos(pitch) * speed;
			dz += -sin(pitch) * speed;
		break;
		case 'a':                 // "A" Влево
			dx += -sin(yaw) * speed;
			dy += cos(yaw) * speed;
			break;
		case 'd':                 // "D" Вправо
			dx += sin(yaw) * speed;
			dy += -cos(yaw) * speed;
		break;
		case 27:
			hipGraphicsUnregisterResource(res);
			glDeleteTextures(2, textures);
			glDeleteBuffers(1, &vbo);
			gluDeleteQuadric(quadratic);
			exit(0);
		break;
	}
}

void mouse(int x, int y) {
	static int x_prev = w / 2, y_prev = h / 2;
	float dx = 0.005 * (x - x_prev);
    float dy = 0.005 * (y - y_prev);
	dyaw -= dx;
    dpitch -= dy;
	x_prev = x;
	y_prev = y;

	// Перемещаем указатель мышки в центр, когда он достиг границы
	if ((x < 20) || (y < 20) || (x > w - 20) || (y > h - 20)) {
		glutWarpPointer(w / 2, h / 2);
		x_prev = w / 2;
		y_prev = h / 2;
    }
}

void reshape(int w_new, int h_new) {
	w = w_new;
	h = h_new;
	glViewport(0, 0, w, h);                                     // Сброс текущей области вывода
	glMatrixMode(GL_PROJECTION);                                // Выбор матрицы проекций
	glLoadIdentity();                                           // Сброс матрицы проекции
}

int main(int argc, char **argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(w, h);
	glutCreateWindow("OpenGL");

	glutIdleFunc(update);
	glutDisplayFunc(display);
	glutKeyboardFunc(keys);
	glutPassiveMotionFunc(mouse);
	glutReshapeFunc(reshape);

	glutSetCursor(GLUT_CURSOR_NONE);	// Скрываем курсор мышки

	int wt, ht;
	FILE *in = fopen("in.data", "rb");
	fread(&wt, sizeof(int), 1, in);
	fread(&ht, sizeof(int), 1, in);
	uchar *data = (uchar *)malloc(sizeof(uchar) * wt * ht * 4);
	fread(data, sizeof(uchar), 4 * wt * ht, in);
	fclose(in);

	glGenTextures(2, textures);
	glBindTexture(GL_TEXTURE_2D, textures[0]);
	glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)wt, (GLsizei)ht, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void*)data);
	// если полигон, на который наносим текстуру, меньше текстуры
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST); //GL_LINEAR);	// Интерполяция
	// если больше
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST); //GL_LINEAR);		
	

	quadratic = gluNewQuadric();
	gluQuadricTexture(quadratic, GL_TRUE);	

	glBindTexture(GL_TEXTURE_2D, textures[1]);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);	// Интерполяция 
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);	// Интерполяция	

	glEnable(GL_TEXTURE_2D);                             // Разрешить наложение текстуры
	glShadeModel(GL_SMOOTH);                             // Разрешение сглаженного закрашивания
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);                // Черный фон
	glClearDepth(1.0f);                                  // Установка буфера глубины
	glDepthFunc(GL_LEQUAL);                              // Тип теста глубины. 
	glEnable(GL_DEPTH_TEST);                			 // Включаем тест глубины
	glEnable(GL_CULL_FACE);                 			 // Режим при котором, тектуры накладываются только с одной стороны

	glewInit();						
	glGenBuffers(1, &vbo);								// Получаем номер буфера
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);			// Делаем его активным
	glBufferData(GL_PIXEL_UNPACK_BUFFER, np * np * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);	// Задаем размер буфера
	hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard);				// Регистрируем буфер для использования его памяти в CUDA
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);			// Деактивируем буфер

	item.x = item.y = item.z = 5.0;						
	item.dx = item.dy = item.dz = 0.1;
	item.q = 1.0;

	glutMainLoop();
}
