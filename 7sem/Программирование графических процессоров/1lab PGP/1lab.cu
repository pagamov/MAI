
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>

__global__ void sub (const double *v1, const double *v2, double *res, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += gridDim.x * blockDim.x) {
        res[i] = v1[i] - v2[i];
    }
}

#define N sizeof(double)*n

int main() {
    int i, n;
    double p;
    std::cin >> n;

    double *arr1 = (double *)malloc(N);
    double *arr2 = (double *)malloc(N);
    double *res =  (double *)malloc(N);

    for (i = 0; i < n; i++) {
        std::cin >> p;
        arr1[i] = p;
    }
    for (i = 0; i < n; i++) {
        std::cin >> p;
        arr2[i] = p;
    }

    double *dev_arr1, *dev_arr2, *dev_res;
    hipMalloc(&dev_arr1, N);
    hipMalloc(&dev_arr2, N);
    hipMalloc(&dev_res,  N);

    hipMemcpy(dev_arr1, arr1, N, hipMemcpyHostToDevice);
    hipMemcpy(dev_arr2, arr2, N, hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, N, hipMemcpyHostToDevice);

    sub <<<2,2>>>(dev_arr1, dev_arr2, dev_res, n);

    hipDeviceSynchronize();

    hipMemcpy(res, dev_res, N, hipMemcpyDeviceToHost);

    for (i = 0; i < n; i++)
        printf("%lf ", res[i]);
    std::cout << std::endl;

    hipFree(dev_arr1);
    hipFree(dev_arr2);
    hipFree(dev_res);

    free(arr1);
    free(arr2);
    free(res);
    return 0;
}
